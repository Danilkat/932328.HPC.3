#include "hip/hip_runtime.h"
#include "./cuda_shared_mmul.cuh"

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include ""
#include "./../type.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "./../cublas_utils.h"

__global__ void gpu_matrix_mult(T* a, T* b, T* c, int m, int n, int k)
{
    //row
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    //col
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    T sum = 0;
    if (j < n && i < m)
    {
        for (int l = 0; l < k; l++)
        {
            sum += a[j + n * l] * b[l + i * k];
        }
        c[i * n + j] = sum;
    }
}

T* executeGPU(int n, int m, int k, T* A, T* B, double* time) {
    T* h_A = A, * h_B = B, * h_C = (T*)malloc(sizeof(T) * m * n);
    int lda = n, ldb = k, ldc = n;

    if (n <= SIZE_LIMIT && k <= SIZE_LIMIT && n <= SIZE_LIMIT) {
        printf("A\n");
        print_matrix<T>(n, k, h_A, lda);
        printf("=====\n");

        printf("B\n");
        print_matrix<T>(k, m, h_B, ldb);
        printf("=====\n");
    }

    const T alpha = 1.0;
    const T beta = 0.0;

    T* d_A = nullptr;
    T* d_B = nullptr;
    T* d_C = nullptr;
    CUDA_CHECK(hipSetDevice(0));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc((void**)(&d_A), sizeof(T) * n * k));
    CUDA_CHECK(hipMalloc((void**)(&d_B), sizeof(T) * k * m));
    CUDA_CHECK(hipMalloc((void**)(&d_C), sizeof(T) * n * m));

    // ����������� � ������� �����������
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(T) * n * k, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(T) * k * m, hipMemcpyHostToDevice));

    /* step 3: compute */
    unsigned int grid_rows = (m + BLOCKSIZE - 1) / BLOCKSIZE;
    unsigned int grid_cols = (k + BLOCKSIZE - 1) / BLOCKSIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
    gpu_matrix_mult << <dimGrid, dimBlock >> > (d_A, d_B, d_C, m, n, k);

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpy(h_C, d_C, sizeof(T) * n * m, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipDeviceSynchronize());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    *time = milliseconds;

    if (n * m <= SIZE_LIMIT * SIZE_LIMIT * OUTPUT_MULTIPLIER) {
        printf("C\n");
        print_matrix(n, m, h_C, ldc);
        printf("=====\n");
    }
    printf("����� ���������� GPU: %f ��.\n\n", milliseconds);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUDA_CHECK(hipDeviceReset());
    return h_C;
}
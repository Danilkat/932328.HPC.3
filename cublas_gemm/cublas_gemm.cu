#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include ""
#include "./cublas_gemm.cuh"
#include "./../cublas_utils.h"


T* generate_random_int(int m, int n, int sigma) {
    T* A = (T*)malloc(sizeof(T) * m * n);
    int drop = 0;
    generate_random_matrix<T>(m, n, &A, &drop);
    for (size_t i = 0; i < m * n; i++)
    {
        T val = round(A[i] * sigma);
        A[i] = (abs(val) > std::numeric_limits<T>::epsilon()) ? val : 0.0;
    };
    if (m * n <= SIZE_LIMIT * SIZE_LIMIT) {
        for (size_t i = 0; i < m*n; i++)
        {
            std::printf("%3.0f ", A[i]);
        }
        std::printf("\n");
    }
    return A;
};

T* executeCublas(int n, int m, int k, double *A, double *B) {
	hipblasHandle_t cublasH = NULL;
	hipStream_t stream = NULL;
	T* h_A = A, * h_B = B, * h_C = (T*)malloc(sizeof(T) * n * m);
	int lda = n, ldb = k, ldc = n;

	if (n <= SIZE_LIMIT && k <= SIZE_LIMIT && n <= SIZE_LIMIT) {
		printf("A\n");
		print_matrix<T>(n, k, h_A, lda);
		printf("=====\n");

		printf("B\n");
		print_matrix<T>(k, m, h_B, ldb);
		printf("=====\n");
	}

	const double alpha = 1.0;
	const double beta = 0.0;

	T* d_A = nullptr;
	T* d_B = nullptr;
	T* d_C = nullptr;

	hipblasOperation_t transa = HIPBLAS_OP_N;
	hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc((void**)(&d_A), sizeof(T) * n * k));
    CUDA_CHECK(hipMalloc((void**)(&d_B), sizeof(T) * k * m));
    CUDA_CHECK(hipMalloc((void**)(&d_C), sizeof(T) * n * m));

    // ����������� � ������� �����������
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    CUDA_CHECK(hipMemcpyAsync(d_A, h_A, sizeof(T) * n * k, hipMemcpyHostToDevice,
        stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, h_B, sizeof(T) * k * m, hipMemcpyHostToDevice,
        stream));

    /* step 3: compute */
    CUBLAS_CHECK(
    hipblasDgemm(cublasH, transa, transb, n, m, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc)
        );

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(h_C, d_C, sizeof(T) * n * m, hipMemcpyDeviceToHost,
        stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    /*
     *   C = | 23.0 | 31.0 |
     *       | 34.0 | 46.0 |
     */
    if (n * m <= SIZE_LIMIT * SIZE_LIMIT * OUTPUT_MULTIPLIER) {
        printf("C\n");
        print_matrix(n, m, h_C, ldc);
        printf("=====\n");
    }
    printf("����� ���������� CUBLAS: %f ��.\n\n", milliseconds);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());
    return h_C;
}